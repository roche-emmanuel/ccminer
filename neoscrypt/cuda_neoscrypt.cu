#include "hip/hip_runtime.h"

/*
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""



#include <stdint.h>
#include <memory.h>
*/
#include <stdio.h>
#include <memory.h>
#include "cuda_vector.h"

// extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);
//hipStream_t stream[4];
#define vectype uintx64bis
#define vectypeS uint28
//#define vectype ulonglong16
//#define vectypeS ulonglong4

#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
#define __LDG_PTR   "l"
#else
#define __LDG_PTR   "r"
#endif


__device__ __align__(16) vectypeS *  W;
__device__  __align__(16) vectypeS * W2;

__device__  vectypeS* Tr;
__device__  vectypeS* Tr2;
__device__ vectypeS* Input;
__device__ vectypeS* B2;

//vectypeS *d_output;
uint32_t *d_NNonce[MAX_GPUS];
uint32_t *d_nnounce[MAX_GPUS];
unsigned long long *d_time[MAX_GPUS];

// Global streams array:
hipStream_t g_stream[MAX_GPUS*2];

__constant__  uint32_t pTarget[8];
__constant__  uint32_t key_init[16];
__constant__  uint32_t input_init[16];
__constant__  uint32_t  c_data[80];
//__constant__  uint8_t  c_data2[320];


#define SALSA_SMALL_UNROLL 1
#define CHACHA_SMALL_UNROLL 1
#define BLAKE2S_BLOCK_SIZE    64U
#define BLAKE2S_OUT_SIZE      32U
#define BLAKE2S_KEY_SIZE      32U
#define BLOCK_SIZE            64U
#define FASTKDF_BUFFER_SIZE  256U
#define PASSWORD_LEN          80U
/// constants ///

static const __constant__  uint8 BLAKE2S_IV_Vec =
{
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};


static const  uint8 BLAKE2S_IV_Vechost =
{
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint32_t BLAKE2S_SIGMA_host[10][16] =
{
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
};

__constant__ uint32_t BLAKE2S_SIGMA[10][16] =
{
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
};

/*
__constant__ uint2 BLAKE2S_SIGMA2[80] =
{
	 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 ,
	 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 ,
	 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 ,
	 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 ,
	 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 ,
	 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 ,
	 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 ,
	 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 ,
	 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 ,
	 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 ,
};
*/


#define SALSA(a,b,c,d) { \
    t =a+d; t=rotate(t,  7);b^=t;    \
    t =b+a; t=rotate(t,  9);c^=t;    \
    t =c+b; t=rotate(t, 13);d^=t;    \
    t =d+c; t=rotate(t, 18);a^=t;     \
}



#define SALSA_CORE(state) { \
\
SALSA(state.s0,state.s4,state.s8,state.sc); \
SALSA(state.s5,state.s9,state.sd,state.s1); \
SALSA(state.sa,state.se,state.s2,state.s6); \
SALSA(state.sf,state.s3,state.s7,state.sb); \
SALSA(state.s0,state.s1,state.s2,state.s3); \
SALSA(state.s5,state.s6,state.s7,state.s4); \
SALSA(state.sa,state.sb,state.s8,state.s9); \
SALSA(state.sf,state.sc,state.sd,state.se); \
	}

static __forceinline__ __device__ void shift256R4(uint32_t * ret, const uint8 &vec4, uint32_t shift2)
{
	uint32_t shift = 32 - shift2;
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[0]) : "r"(0), "r"(vec4.s0), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[1]) : "r"(vec4.s0), "r"(vec4.s1), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[2]) : "r"(vec4.s1), "r"(vec4.s2), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[3]) : "r"(vec4.s2), "r"(vec4.s3), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[4]) : "r"(vec4.s3), "r"(vec4.s4), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[5]) : "r"(vec4.s4), "r"(vec4.s5), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[6]) : "r"(vec4.s5), "r"(vec4.s6), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[7]) : "r"(vec4.s6), "r"(vec4.s7), "r"(shift));
	asm("shr.b32         %0, %1, %2;"     : "=r"(ret[8]) : "r"(vec4.s7), "r"(shift));


}

static __device__ __inline__ void chacha_step(uint32_t &a, uint32_t &b, uint32_t &c, uint32_t &d)
{
	asm("{\n\t"
	    "add.u32 %0,%0,%1; \n\t"
	    "xor.b32 %3,%3,%0; \n\t"
	    "prmt.b32 %3, %3, 0, 0x1032; \n\t"
	    "add.u32 %2,%2,%3; \n\t"
	    "xor.b32 %1,%1,%2; \n\t"
	    "shf.l.wrap.b32 %1, %1, %1, 12; \n\t"
	    "add.u32 %0,%0,%1; \n\t"
	    "xor.b32 %3,%3,%0; \n\t"
	    "prmt.b32 %3, %3, 0, 0x2103; \n\t"
	    "add.u32 %2,%2,%3; \n\t"
	    "xor.b32 %1,%1,%2; \n\t"
	    "shf.l.wrap.b32 %1, %1, %1, 7; \n\t}"
	    : "+r"(a), "+r"(b), "+r"(c), "+r"(d));
}

static __device__ __inline__ void chacha_step3(uint32_t* ptr)
{
	asm volatile ("{\n\t"
		".reg .u32 s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15; \n\t"
		"ld.v4.u32 {s0,s1,s2,s3}, [%0]; \n\t"
		"ld.v4.u32 {s4,s5,s6,s7}, [%0+16]; \n\t"
		"ld.v4.u32 {s8,s9,s10,s11}, [%0+32]; \n\t"
		"ld.v4.u32 {s12,s13,s14,s15}, [%0+48]; \n\t"

		// %0 -> s0
		// %1 -> s4
		// %2 -> s8
		// %3 -> s12
    "add.u32 s0,s0,s4; \n\t"
    "xor.b32 s12,s12,s0; \n\t"
    "prmt.b32 s12, s12, 0, 0x1032; \n\t"
    "add.u32 s8,s8,s12; \n\t"
    "xor.b32 s4,s4,s8; \n\t"
    "shf.l.wrap.b32 s4, s4, s4, 12; \n\t"
    "add.u32 s0,s0,s4; \n\t"
    "xor.b32 s12,s12,s0; \n\t"
    "prmt.b32 s12, s12, 0, 0x2103; \n\t"
    "add.u32 s8,s8,s12; \n\t"
    "xor.b32 s4,s4,s8; \n\t"
    "shf.l.wrap.b32 s4, s4, s4, 7; \n\t"


    "st.v4.u32  [%0], {s0,s1,s2,s3}; \n\t"
    "st.v4.u32  [%0+16], {s4,s5,s6,s7}; \n\t"
    "st.v4.u32  [%0+32], {s8,s9,s10,s11}; \n\t"
    "st.v4.u32  [%0+48], {s12,s13,s14,s15}; \n\t"
    "}"
    :: "l"(ptr));
}

static __device__ __inline__ void chacha_step4(uint32_t* X)
{
	asm volatile ("{\n\t"

		// %0 -> %0
		// %1 -> %4
		// %2 -> %8
		// %3 -> %12
    "add.u32 %0,%0,%4; \n\t"
    "xor.b32 %12,%12,%0; \n\t"
    "prmt.b32 %12, %12, 0, 0x1032; \n\t"
    "add.u32 %8,%8,%12; \n\t"
    "xor.b32 %4,%4,%8; \n\t"
    "shf.l.wrap.b32 %4, %4, %4, 12; \n\t"
    "add.u32 %0,%0,%4; \n\t"
    "xor.b32 %12,%12,%0; \n\t"
    "prmt.b32 %12, %12, 0, 0x2103; \n\t"
    "add.u32 %8,%8,%12; \n\t"
    "xor.b32 %4,%4,%8; \n\t"
    "shf.l.wrap.b32 %4, %4, %4, 7; \n\t"

		// %0 -> %1
		// %1 -> %5
		// %2 -> %9
		// %3 -> %13
    "add.u32 %1,%1,%5; \n\t"
    "xor.b32 %13,%13,%1; \n\t"
    "prmt.b32 %13, %13, 0, 0x1032; \n\t"
    "add.u32 %9,%9,%13; \n\t"
    "xor.b32 %5,%5,%9; \n\t"
    "shf.l.wrap.b32 %5, %5, %5, 12; \n\t"
    "add.u32 %1,%1,%5; \n\t"
    "xor.b32 %13,%13,%1; \n\t"
    "prmt.b32 %13, %13, 0, 0x2103; \n\t"
    "add.u32 %9,%9,%13; \n\t"
    "xor.b32 %5,%5,%9; \n\t"
    "shf.l.wrap.b32 %5, %5, %5, 7; \n\t"

		// %0 -> %2
		// %1 -> %6
		// %2 -> %10
		// %3 -> %14
    "add.u32 %2,%2,%6; \n\t"
    "xor.b32 %14,%14,%2; \n\t"
    "prmt.b32 %14, %14, 0, 0x1032; \n\t"
    "add.u32 %10,%10,%14; \n\t"
    "xor.b32 %6,%6,%10; \n\t"
    "shf.l.wrap.b32 %6, %6, %6, 12; \n\t"
    "add.u32 %2,%2,%6; \n\t"
    "xor.b32 %14,%14,%2; \n\t"
    "prmt.b32 %14, %14, 0, 0x2103; \n\t"
    "add.u32 %10,%10,%14; \n\t"
    "xor.b32 %6,%6,%10; \n\t"
    "shf.l.wrap.b32 %6, %6, %6, 7; \n\t"

		// %0 -> %3
		// %1 -> %7
		// %2 -> %11
		// %3 -> %15
    "add.u32 %3,%3,%7; \n\t"
    "xor.b32 %15,%15,%3; \n\t"
    "prmt.b32 %15, %15, 0, 0x1032; \n\t"
    "add.u32 %11,%11,%15; \n\t"
    "xor.b32 %7,%7,%11; \n\t"
    "shf.l.wrap.b32 %7, %7, %7, 12; \n\t"
    "add.u32 %3,%3,%7; \n\t"
    "xor.b32 %15,%15,%3; \n\t"
    "prmt.b32 %15, %15, 0, 0x2103; \n\t"
    "add.u32 %11,%11,%15; \n\t"
    "xor.b32 %7,%7,%11; \n\t"
    "shf.l.wrap.b32 %7, %7, %7, 7; \n\t"

		// %0 -> %0
		// %1 -> %5
		// %2 -> %10
		// %3 -> %15
    "add.u32 %0,%0,%5; \n\t"
    "xor.b32 %15,%15,%0; \n\t"
    "prmt.b32 %15, %15, 0, 0x1032; \n\t"
    "add.u32 %10,%10,%15; \n\t"
    "xor.b32 %5,%5,%10; \n\t"
    "shf.l.wrap.b32 %5, %5, %5, 12; \n\t"
    "add.u32 %0,%0,%5; \n\t"
    "xor.b32 %15,%15,%0; \n\t"
    "prmt.b32 %15, %15, 0, 0x2103; \n\t"
    "add.u32 %10,%10,%15; \n\t"
    "xor.b32 %5,%5,%10; \n\t"
    "shf.l.wrap.b32 %5, %5, %5, 7; \n\t"

		// %0 -> %1
		// %1 -> %6
		// %2 -> %11
		// %3 -> %12
    "add.u32 %1,%1,%6; \n\t"
    "xor.b32 %12,%12,%1; \n\t"
    "prmt.b32 %12, %12, 0, 0x1032; \n\t"
    "add.u32 %11,%11,%12; \n\t"
    "xor.b32 %6,%6,%11; \n\t"
    "shf.l.wrap.b32 %6, %6, %6, 12; \n\t"
    "add.u32 %1,%1,%6; \n\t"
    "xor.b32 %12,%12,%1; \n\t"
    "prmt.b32 %12, %12, 0, 0x2103; \n\t"
    "add.u32 %11,%11,%12; \n\t"
    "xor.b32 %6,%6,%11; \n\t"
    "shf.l.wrap.b32 %6, %6, %6, 7; \n\t"

		// %0 -> %2
		// %1 -> %7
		// %2 -> %8
		// %3 -> %13
    "add.u32 %2,%2,%7; \n\t"
    "xor.b32 %13,%13,%2; \n\t"
    "prmt.b32 %13, %13, 0, 0x1032; \n\t"
    "add.u32 %8,%8,%13; \n\t"
    "xor.b32 %7,%7,%8; \n\t"
    "shf.l.wrap.b32 %7, %7, %7, 12; \n\t"
    "add.u32 %2,%2,%7; \n\t"
    "xor.b32 %13,%13,%2; \n\t"
    "prmt.b32 %13, %13, 0, 0x2103; \n\t"
    "add.u32 %8,%8,%13; \n\t"
    "xor.b32 %7,%7,%8; \n\t"
    "shf.l.wrap.b32 %7, %7, %7, 7; \n\t"

		// %0 -> %3
		// %1 -> %4
		// %2 -> %9
		// %3 -> %14
    "add.u32 %3,%3,%4; \n\t"
    "xor.b32 %14,%14,%3; \n\t"
    "prmt.b32 %14, %14, 0, 0x1032; \n\t"
    "add.u32 %9,%9,%14; \n\t"
    "xor.b32 %4,%4,%9; \n\t"
    "shf.l.wrap.b32 %4, %4, %4, 12; \n\t"
    "add.u32 %3,%3,%4; \n\t"
    "xor.b32 %14,%14,%3; \n\t"
    "prmt.b32 %14, %14, 0, 0x2103; \n\t"
    "add.u32 %9,%9,%14; \n\t"
    "xor.b32 %4,%4,%9; \n\t"
    "shf.l.wrap.b32 %4, %4, %4, 7; \n\t"

    "}"
    : "+r"(X[0]), "+r"(X[1]), "+r"(X[2]), "+r"(X[3]), 
      "+r"(X[4]), "+r"(X[5]), "+r"(X[6]), "+r"(X[7]),
      "+r"(X[8]), "+r"(X[9]), "+r"(X[10]),"+r"(X[11]),
      "+r"(X[12]),"+r"(X[13]),"+r"(X[14]),"+r"(X[15])); //, "+r"(X[4]), "+r"(X[5]), "+r"(X[6]), "+r"(X[7])
}
static __device__ __inline__ void chacha_step2(uint32_t &a, uint32_t &b, uint32_t &c, uint32_t &d,
																							 uint32_t &e, uint32_t &f, uint32_t &g, uint32_t &h)
{
	asm("{\n\t"
			".reg .u r0,r1,r2,r3; \n\t"
			"mov.b64 r0, {%0,%4}; \n\t"
			"mov.b64 r1, {%1,%5}; \n\t"
			"mov.b64 r2, {%2,%6}; \n\t"
			"mov.b64 r3, {%3,%7}; \n\t"

			"add.u64 r0,r0,r1; \n\t"
	    "xor.b64 r3,r3,r0; \n\t"
	    "prmt.b64 r3, r3, 0, 0x1032; \n\t"


	    "add.u32 %0,%0,%1; \n\t"
	    "xor.b32 %3,%3,%0; \n\t"
	    "prmt.b32 %3, %3, 0, 0x1032; \n\t"
	    "add.u32 %2,%2,%3; \n\t"
	    "xor.b32 %1,%1,%2; \n\t"
	    "shf.l.wrap.b32 %1, %1, %1, 12; \n\t"
	    "add.u32 %0,%0,%1; \n\t"
	    "xor.b32 %3,%3,%0; \n\t"
	    "prmt.b32 %3, %3, 0, 0x2103; \n\t"
	    "add.u32 %2,%2,%3; \n\t"
	    "xor.b32 %1,%1,%2; \n\t"
	    "shf.l.wrap.b32 %1, %1, %1, 7; \n\t"

	    "add.u32 %4,%4,%5; \n\t"
	    "xor.b32 %7,%7,%4; \n\t"
	    "prmt.b32 %7, %7, 0, 0x1032; \n\t"
	    "add.u32 %6,%6,%7; \n\t"
	    "xor.b32 %5,%5,%6; \n\t"
	    "shf.l.wrap.b32 %5, %5, %5, 12; \n\t"
	    "add.u32 %4,%4,%5; \n\t"
	    "xor.b32 %7,%7,%4; \n\t"
	    "prmt.b32 %7, %7, 0, 0x2103; \n\t"
	    "add.u32 %6,%6,%7; \n\t"
	    "xor.b32 %5,%5,%6; \n\t"
	    "shf.l.wrap.b32 %5, %5, %5, 7; \n\t"


	    "}"
	    : "+r"(a), "+r"(b), "+r"(c), "+r"(d), "+r"(e), "+r"(f), "+r"(g), "+r"(h));
}

#if __CUDA_ARCH__ >=500

#define CHACHA_STEP(a,b,c,d) { \
a += b; d = __byte_perm(d^a,0,0x1032); \
c += d; b = rotate(b^c, 12); \
a += b; d = __byte_perm(d^a,0,0x2103); \
c += d; b = rotate(b^c, 7); \
	}

//#define CHACHA_STEP(a,b,c,d) chacha_step(a,b,c,d)
#else
#define CHACHA_STEP(a,b,c,d) { \
a += b; d = rotate(d^a,16); \
c += d; b = rotate(b^c, 12); \
a += b; d = rotate(d^a,8); \
c += d; b = rotate(b^c, 7); \
	}
#endif

#define CHACHA_CORE_PARALLEL(state)	 { \
 \
  chacha_step(state.lo.s0, state.lo.s4, state.hi.s0, state.hi.s4); \
  chacha_step(state.lo.s1, state.lo.s5, state.hi.s1, state.hi.s5); \
  chacha_step(state.lo.s2, state.lo.s6, state.hi.s2, state.hi.s6); \
	chacha_step(state.lo.s3, state.lo.s7, state.hi.s3, state.hi.s7); \
	chacha_step(state.lo.s0, state.lo.s5, state.hi.s2, state.hi.s7); \
  chacha_step(state.lo.s1, state.lo.s6, state.hi.s3, state.hi.s4); \
  chacha_step(state.lo.s2, state.lo.s7, state.hi.s0, state.hi.s5); \
	chacha_step(state.lo.s3, state.lo.s4, state.hi.s1, state.hi.s6); \
\
}

  // chacha_step(state.lo.s2, state.lo.s7, state.hi.s0, state.hi.s5); \
	// chacha_step(state.lo.s3, state.lo.s4, state.hi.s1, state.hi.s6); \

#define CHACHA_CORE_PARALLEL_B(state)	 { \
 \
  chacha_step4((uint32_t*)&state); \
\
}

// #define CHACHA_CORE_PARALLEL_B(state)	 { \
//  \
//   chacha_step2(state.lo.s0, state.lo.s4, state.hi.s0, state.hi.s4, state.lo.s1, state.lo.s5, state.hi.s1, state.hi.s5); \
//   chacha_step2(state.lo.s2, state.lo.s6, state.hi.s2, state.hi.s6, state.lo.s3, state.lo.s7, state.hi.s3, state.hi.s7); \
// 	chacha_step2(state.lo.s0, state.lo.s5, state.hi.s2, state.hi.s7, state.lo.s1, state.lo.s6, state.hi.s3, state.hi.s4); \
//   chacha_step2(state.lo.s2, state.lo.s7, state.hi.s0, state.hi.s5, state.lo.s3, state.lo.s4, state.hi.s1, state.hi.s6); \
// \
// }

#define CHACHA_CORE_PARALLEL2(i0,state)	 { \
 \
    CHACHA_STEP(state[2*i0].x.x, state[2*i0].z.x, state[2*i0+1].x.x, state[2*i0+1].z.x); \
    CHACHA_STEP(state[2*i0].x.y, state[2*i0].z.y, state[2*i0+1].x.y, state[2*i0+1].z.y); \
    CHACHA_STEP(state[2*i0].y.x, state[2*i0].w.x, state[2*i0+1].y.x, state[2*i0+1].w.x); \
	CHACHA_STEP(state[2*i0].y.y, state[2*i0].w.y, state[2*i0+1].y.y, state[2*i0+1].w.y); \
	CHACHA_STEP(state[2*i0].x.x, state[2*i0].z.y, state[2*i0+1].y.x, state[2*i0+1].w.y); \
    CHACHA_STEP(state[2*i0].x.y, state[2*i0].w.x, state[2*i0+1].y.y, state[2*i0+1].z.x); \
    CHACHA_STEP(state[2*i0].y.x, state[2*i0].w.y, state[2*i0+1].x.x, state[2*i0+1].z.y); \
	CHACHA_STEP(state[2*i0].y.y, state[2*i0].z.x, state[2*i0+1].x.y, state[2*i0+1].w.x); \
\
	}



// Blake2S

#define BLAKE2S_BLOCK_SIZE    64U
#define BLAKE2S_OUT_SIZE      32U
#define BLAKE2S_KEY_SIZE      32U

#if __CUDA_ARCH__ >= 500
#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
    a += b; d = __byte_perm(d^a,0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
idx = BLAKE2S_SIGMA[idx0][idx1+1]; a += key[idx]; \
    a += b; d = __byte_perm(d^a,0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
	}
#else
#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
    a += b; d = rotate(d^a,16); \
	c += d; b = rotateR(b^c, 12); \
idx = BLAKE2S_SIGMA[idx0][idx1+1]; a += key[idx]; \
    a += b; d = rotateR(d^a,8); \
	c += d; b = rotateR(b^c, 7); \
		}
#endif

#if __CUDA_ARCH__ >= 500


#define BLAKE(a, b, c, d, key1,key2) { \
   \
    a += key1; \
    a += b; d = __byte_perm(d^a,0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
    a += key2; \
    a += b; d = __byte_perm(d^a,0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
	 	 }

#define BLAKE_G_PRE(idx0,idx1, a, b, c, d, key) { \
    a += key[idx0]; \
    a += b; d = __byte_perm(d^a,0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
    a += key[idx1]; \
    a += b; d = __byte_perm(d^a,0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
		}

#define BLAKE_G_PRE0(idx0,idx1, a, b, c, d, key) { \
    \
    a += b; d = __byte_perm(d^a,0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
    \
    a += b; d = __byte_perm(d^a,0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
				}

#define BLAKE_G_PRE1(idx0,idx1, a, b, c, d, key) { \
    a += key[idx0]; \
    a += b; d = __byte_perm(d^a,0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
    \
    a += b; d = __byte_perm(d^a,0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
				}

#define BLAKE_G_PRE2(idx0,idx1, a, b, c, d, key) { \
    \
    a += b; d = __byte_perm(d^a,0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
    a += key[idx1]; \
    a += b; d = __byte_perm(d^a,0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
				}

#else
#define BLAKE(a, b, c, d, key1,key2) { \
  \
    a += key1; \
    a += b; d = rotate(d^a,16); \
	c += d; b = rotateR(b^c, 12); \
    a += key2; \
    a += b; d = rotateR(d^a,8); \
	c += d; b = rotateR(b^c, 7); \
	}


#define BLAKE_G_PRE(idx0, idx1, a, b, c, d, key) { \
    a += key[idx0]; \
    a += b; d = rotate(d^a,16); \
	c += d; b = rotateR(b^c, 12); \
    a += key[idx1]; \
    a += b; d = rotateR(d^a,8); \
	c += d; b = rotateR(b^c, 7); \
				}

#define BLAKE_G_PRE0(idx0, idx1, a, b, c, d, key) { \
     \
    a += b; d = rotate(d^a,16); \
	c += d; b = rotateR(b^c, 12); \
    \
    a += b; d = rotateR(d^a,8); \
	c += d; b = rotateR(b^c, 7); \
								}

#define BLAKE_G_PRE1(idx0, idx1, a, b, c, d, key) { \
    a += key[idx0]; \
    a += b; d = rotate(d^a,16); \
	c += d; b = rotateR(b^c, 12); \
    a += b; d = rotateR(d^a,8); \
	c += d; b = rotateR(b^c, 7); \
								}

#define BLAKE_G_PRE2(idx0, idx1, a, b, c, d, key) { \
     \
    a += b; d = rotate(d^a,16); \
	c += d; b = rotateR(b^c, 12); \
    a += key[idx1]; \
    a += b; d = rotateR(d^a,8); \
	c += d; b = rotateR(b^c, 7); \
								}


#endif




#define BLAKE_Ghost(idx0, idx1, a, b, c, d, key) { \
idx = BLAKE2S_SIGMA_host[idx0][idx1]; a += key[idx]; \
    a += b; d = ROTR32(d^a,16); \
	c += d; b = ROTR32(b^c, 12); \
idx = BLAKE2S_SIGMA_host[idx0][idx1+1]; a += key[idx]; \
    a += b; d = ROTR32(d^a,8); \
	c += d; b = ROTR32(b^c, 7); \
		}


static __forceinline__ __device__ void Blake2S(uint32_t *out, const uint32_t* __restrict__  inout, const  uint32_t * __restrict__ TheKey)
{
	uint16 V;

	uint32_t idx;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	V.lo = BLAKE2S_IV_Vec;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;


	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


//		{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE0(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


//		{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE0(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


//		{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE1(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE1(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);



//		{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE2(9, 0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(5, 7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(10, 15, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(14, 1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(11, 12, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(3, 13, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//		{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE1(2, 12, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 10, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE1(0, 11, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(8, 3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(4, 13, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(7, 5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(15, 14, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(1, 9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


//		{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE2(12, 5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(1, 15, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 13, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(4, 10, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(6, 3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(9, 2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(8, 11, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


//		{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE0(13, 11, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(7, 14, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(12, 1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(3, 9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(15, 4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 10, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//		{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE1(6, 15, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE0(14, 9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(11, 3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(0, 8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(12, 2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(13, 7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(10, 5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//		{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE2(10, 2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(7, 6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(1, 5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(15, 11, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(9, 14, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(3, 12, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(13, 0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	V.lo ^= V.hi;
	V.lo ^= tmpblock;


	V.hi = BLAKE2S_IV_Vec;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);


	//		{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	//		{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);


	//		{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);


	for (int x = 4; x < 10; ++x)
	{
		BLAKE_G(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_G(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_G(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_G(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_G(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_G(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_G(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_G(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	((uint8*)out)[0] = V.lo;

}

static __forceinline__ __device__ void Blake2S_v2(uint32_t *out, const uint32_t* __restrict__  inout, const  uint32_t * __restrict__ TheKey)
{
	uint16 V;

	uint2 idx;
	uint8 tmpblock;
	//	uint16 inout[1];
	//	inout[0] = ((uint16*)inoutE)[0];

	V.hi = BLAKE2S_IV_Vec;
	V.lo = BLAKE2S_IV_Vec;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;


	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


	//		{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE0(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


	//		{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE0(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


	//		{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE1(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE1(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);



	//		{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE2(9, 0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(5, 7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(10, 15, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(14, 1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(11, 12, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(3, 13, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	//		{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE1(2, 12, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 10, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE1(0, 11, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(8, 3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(4, 13, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(7, 5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(15, 14, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(1, 9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


	//		{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE2(12, 5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(1, 15, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 13, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(4, 10, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(6, 3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(9, 2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(8, 11, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);


	//		{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE0(13, 11, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(7, 14, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(12, 1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(3, 9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(15, 4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 10, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	//		{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE1(6, 15, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE0(14, 9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(11, 3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(0, 8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(12, 2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(13, 7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(10, 5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	//		{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE2(10, 2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(7, 6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(1, 5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(15, 11, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(9, 14, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(3, 12, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(13, 0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	V.lo ^= V.hi;
	V.lo ^= tmpblock;


	V.hi = BLAKE2S_IV_Vec;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);


	//		{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	//		{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);


	//		{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	//#pragma unroll

	//		13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10,
	//		6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5,
	//		10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0,

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[9], inout[0]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[5], inout[7]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[2], inout[4]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[10], inout[15]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[14], inout[1]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[11], inout[12]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[6], inout[8]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[3], inout[13]);

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[2], inout[12]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[6], inout[10]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[0], inout[11]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[8], inout[3]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[4], inout[13]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[7], inout[5]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[15], inout[14]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[1], inout[9]);

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[12], inout[5]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[1], inout[15]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[14], inout[13]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[4], inout[10]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[0], inout[7]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[6], inout[3]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[9], inout[2]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[8], inout[11]);

	//		13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10,
	//		6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5,

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[13], inout[11]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[7], inout[14]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[12], inout[1]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[3], inout[9]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[5], inout[0]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[15], inout[4]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[8], inout[6]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[2], inout[10]);

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[6], inout[15]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[14], inout[9]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[11], inout[3]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[0], inout[8]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[12], inout[2]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[13], inout[7]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[1], inout[4]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[10], inout[5]);
	//		10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0,
	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[10], inout[2]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[8], inout[4]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[7], inout[6]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[1], inout[5]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[15], inout[11]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[9], inout[14]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[3], inout[12]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[13], inout[0]);




	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	((uint8*)out)[0] = V.lo;

}


static __forceinline__ __device__ uint16 salsa_small_scalar_rnd(const uint16 &X)
{
	uint16 state = X;
	uint32_t t;

	#pragma unroll 1
	for (int i = 0; i < 10; ++i) { SALSA_CORE(state);}

	return (X + state);
}

static __device__ __forceinline__ uint16 chacha_small_parallel_rnd(const uint16 &X)
{
	uint16 st = X;
	#pragma nounroll
	for (int i = 0; i < 10; ++i) {
		CHACHA_CORE_PARALLEL_B(st);
	}
	return (X + st);
}


static __device__ __forceinline__ void neoscrypt_chacha(uint16 *XV)
{
	XV[0] ^= XV[3];
	uint16 temp;

	XV[0] = chacha_small_parallel_rnd(XV[0]); 
	XV[1] ^= XV[0];
	temp = chacha_small_parallel_rnd(XV[1]); 
	XV[2] ^= temp;
	XV[1] = chacha_small_parallel_rnd(XV[2]); 
	XV[3] ^= XV[1];
	XV[3] = chacha_small_parallel_rnd(XV[3]);
	XV[2] = temp;
}

static __device__ __forceinline__ void neoscrypt_salsa(uint16 *XV)
{

	XV[0] ^= XV[3];
	uint16 temp;

	XV[0] = salsa_small_scalar_rnd(XV[0]);
	XV[1] ^= XV[0];
	temp = salsa_small_scalar_rnd(XV[1]);
	XV[2] ^= temp;
	XV[1] = salsa_small_scalar_rnd(XV[2]);
	XV[3] ^= XV[1];
	XV[3] = salsa_small_scalar_rnd(XV[3]);
	XV[2] = temp;
}



static __forceinline__ __host__ void Blake2Shost(uint32_t * inout, const uint32_t * inkey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;



	V.hi = BLAKE2S_IV_Vechost;
	V.lo = BLAKE2S_IV_Vechost;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inkey);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inkey);
	}

	V.lo ^= V.hi;
	V.lo ^= tmpblock;


	V.hi = BLAKE2S_IV_Vechost;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	V.lo ^= V.hi ^ tmpblock;

	((uint8*)inout)[0] = V.lo;
}

static __forceinline__ __device__ void fastkdf256_v1(int thread, const uint32_t nonce, const uint32_t * __restrict__  s_data) //, vectypeS * output)
{
	vectypeS output[8];
	uint8_t bufidx = 0;
	uchar4 bufhelper;
	uint32_t data18 = s_data[18];
	uint32_t data20 = s_data[0];
	uint32_t B[64];

	((uintx64*)(B))[0] = ((uintx64*)s_data)[0];
	((uint32_t*)B)[19] = nonce;
	((uint32_t*)B)[39] = nonce;
	((uint32_t*)B)[59] = nonce;

	uint32_t input[BLAKE2S_BLOCK_SIZE / 4]; uint32_t key[BLAKE2S_BLOCK_SIZE / 4] = { 0 };

	((uint816*)input)[0] = ((uint816*)input_init)[0];
	((uint48*)key)[0] = ((uint48*)key_init)[0];

	uint32_t qbuf, rbuf, bitbuf;

#pragma unroll  1
	for (int i = 0; i < 31; ++i)
	{

		bufhelper = ((uchar4*)input)[0];
		for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
		bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;

		qbuf = bufidx / 4;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;


		uint32_t shifted[9];

		shift256R4(shifted, ((uint8*)input)[0], bitbuf);

		//#pragma unroll
		uint32_t temp[9];

		for (int k = 0; k < 9; ++k) {
			uint32_t indice = (k + qbuf) & 63;
			temp[k] = ((uint32_t*)B)[indice];
			temp[k] ^= shifted[k];
			((uint32_t*)B)[indice] = temp[k];
		}


		uint32_t a = ((uint32_t*)s_data)[qbuf % 64], b;
		//#pragma unroll
		for (int k = 0; k < 8; k++) {
			b = s_data[(qbuf + 2 * k + 1) % 64];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[2 * k]) : "r"(a), "r"(b), "r"(bitbuf));
			a = s_data[(qbuf + 2 * k + 2) % 64];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[2 * k + 1]) : "r"(b), "r"(a), "r"(bitbuf));
		}


//               #pragma unroll
//				for (int k = 0; k<16; k++)
//					asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k]) : "r"(((uint32_t*)s_data)[qbuf + k]), "r"(((uint32_t*)s_data)[qbuf + k + 1]), "r"(bitbuf));

		uint32_t noncepos = 19 - qbuf % 20;
		if (noncepos <= 16 && qbuf < 60) {
			if (noncepos != 0)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos - 1]) : "r"(data18), "r"(nonce), "r"(bitbuf));
			if (noncepos != 16)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos]) : "r"(nonce), "r"(data20), "r"(bitbuf));
		}

		for (int k = 0; k < 8; k++)
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[k]) : "r"(temp[k]), "r"(temp[k + 1]), "r"(bitbuf));

		Blake2S(input, input, key); //yeah right...

	}
	bufhelper = ((uchar4*)input)[0];
	for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
	bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;

	qbuf = bufidx / 4;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;

	for (int i = 0; i < 64; i++)
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(((uint32_t*)output)[i]) : "r"(((uint32_t*)(B))[(qbuf + i)%64]), "r"(((uint32_t*)(B))[(qbuf + i + 1)%64]), "r"(bitbuf));


	//for (int i=0;i<8;i++)
	((ulonglong4*)output)[0] ^= ((ulonglong4*)input)[0];

	((uintx64*)output)[0] ^= ((uintx64*)s_data)[0];
	((uint32_t*)output)[19] ^= nonce;
	((uint32_t*)output)[39] ^= nonce;
	((uint32_t*)output)[59] ^= nonce;


	for (int i = 0; i < 8; i++)
		(Input + 8 * thread)[i] = output[i];

}

static __forceinline__ __device__ void fastkdf256_v2(int thread, const uint32_t nonce, const  uint32_t* __restrict__ s_data) //, vectypeS * output)
{
	vectypeS output[8];
	uint8_t bufidx = 0;
	uchar4 bufhelper;
	uint32_t data18 = s_data[18];
	uint32_t data20 = s_data[0];
#define Bshift 16*thread

	uint32_t* B = (uint32_t*)&B2[Bshift];
	((uintx64*)(B))[0] = ((uintx64*)s_data)[0];


	((uint32_t*)B)[19] = nonce;
	((uint32_t*)B)[39] = nonce;
	((uint32_t*)B)[59] = nonce;
	uint32_t input[16];
	uint32_t key[16] = { 0 };

	((ulonglong4*)input)[0] = ((ulonglong4*)input_init)[0];
	((uint28*)key)[0] = ((uint28*)key_init)[0];

	uint32_t qbuf, rbuf, bitbuf;

#pragma unroll  1
	for (int i = 0; i < 31; ++i)
	{

		bufhelper = ((uchar4*)input)[0];
		for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
		bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;

		qbuf = bufidx / 4;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;
		uint32_t shifted[9];

		shift256R4(shifted, ((uint8*)input)[0], bitbuf);

		//#pragma unroll
		uint32_t temp[9];


		for (int k = 0; k < 9; ++k)
			temp[k] = __ldg(&B[(k + qbuf) & 63]);

		for (int k = 0; k < 9; ++k)
			temp[k] ^= shifted[k];



		uint32_t a = s_data[qbuf % 64], b;
		//#pragma unroll

		for (int k = 0; k < 8; k++) {
			b = s_data[(qbuf + 2 * k + 1) % 64];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[2 * k]) : "r"(a), "r"(b), "r"(bitbuf));
			a = s_data[(qbuf + 2 * k + 2) % 64];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[2 * k + 1]) : "r"(b), "r"(a), "r"(bitbuf));
		}


		uint32_t noncepos = 19 - qbuf % 20;
		if (noncepos <= 16 && qbuf < 60) {
			if (noncepos != 0)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos - 1]) : "r"(data18), "r"(nonce), "r"(bitbuf));
			if (noncepos != 16)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos]) : "r"(nonce), "r"(data20), "r"(bitbuf));
		}


		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[0]) : "r"(temp[0]), "r"(temp[1]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[1]) : "r"(temp[1]), "r"(temp[2]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[2]) : "r"(temp[2]), "r"(temp[3]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[3]) : "r"(temp[3]), "r"(temp[4]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[4]) : "r"(temp[4]), "r"(temp[5]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[5]) : "r"(temp[5]), "r"(temp[6]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[6]) : "r"(temp[6]), "r"(temp[7]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[7]) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));


		Blake2S_v2(input, input, key);

		for (int k = 0; k < 9; k++)
			B[(k + qbuf) & 63] = temp[k];

	}

	bufhelper = ((uchar4*)input)[0];
	for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
	bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;

	qbuf = bufidx / 4;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;

	for (int i = 0; i < 64; i++) {
		uint32_t a = (qbuf + i) & 63, b = (qbuf + i + 1) & 63;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(((uint32_t*)output)[i]) : "r"(__ldg(&B[a])), "r"(__ldg(&B[b])), "r"(bitbuf));
	}



	output[0] ^= ((uint28*)input)[0];
	for (int i = 0; i < 8; i++) output[i] ^= ((uint28*)s_data)[i];
//	((ulonglong16 *)output)[0] ^= ((ulonglong16*)s_data)[0];
	((uint32_t*)output)[19] ^= nonce;
	((uint32_t*)output)[39] ^= nonce;
	((uint32_t*)output)[59] ^= nonce;;
	((ulonglong16 *)(Input + 8 * thread))[0] = ((ulonglong16*)output)[0];


}

static __forceinline__ __device__ void fastkdf32_v1(int thread, const  uint32_t  nonce, const uint32_t * __restrict__ salt, const uint32_t * __restrict__  s_data, uint32_t &output)
{



	uint8_t bufidx = 0;
	uchar4 bufhelper;

	uint32_t temp[9];

//	uint32_t  B0[64];
#define Bshift 16*thread

	uint32_t* B0 = (uint32_t*)&B2[Bshift];
	uint32_t cdata7 = s_data[7];
	uint32_t data18 = s_data[18];
	uint32_t data20 = s_data[0];


	((uintx64*)B0)[0] = ((uintx64*)salt)[0];
	uint32_t input[BLAKE2S_BLOCK_SIZE / 4]; uint32_t key[BLAKE2S_BLOCK_SIZE / 4] = { 0 };
	((uint816*)input)[0] = ((uint816*)s_data)[0];
	((uint48*)key)[0] = ((uint48*)salt)[0];
	uint32_t qbuf, rbuf, bitbuf;

#pragma nounroll
	for (int i = 0; i < 31; i++)
	{
#if __CUDA_ARCH__ < 500
		Blake2S(input, input, key);
#else
		Blake2S_v2(input, input, key);
#endif
		bufidx = 0;
		bufhelper = ((uchar4*)input)[0];
		for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
		bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;
		qbuf = bufidx / 4;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;
		uint32_t shifted[9];

		shift256R4(shifted, ((uint8*)input)[0], bitbuf);

		for (int k = 0; k < 9; k++) {
			temp[k] = ((uint32_t *)B0)[(k + qbuf) % 64];
		}

		((uint28*)temp)[0] ^= ((uint28*)shifted)[0];
		temp[8] ^= shifted[8];




		uint32_t a = s_data[qbuf % 64], b;
		//#pragma unroll
		for (int k = 0; k < 8; k++) {
			b = s_data[(qbuf + 2 * k + 1) % 64];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[2 * k]) : "r"(a), "r"(b), "r"(bitbuf));
			a = s_data[(qbuf + 2 * k + 2) % 64];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[2 * k + 1]) : "r"(b), "r"(a), "r"(bitbuf));
		}



		uint32_t noncepos = 19 - qbuf % 20;
		if (noncepos <= 16 && qbuf < 60) {
			if (noncepos != 0)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos - 1]) : "r"(data18), "r"(nonce), "r"(bitbuf));
			if (noncepos != 16)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos]) : "r"(nonce), "r"(data20), "r"(bitbuf));
		}
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[0]) : "r"(temp[0]), "r"(temp[1]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[1]) : "r"(temp[1]), "r"(temp[2]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[2]) : "r"(temp[2]), "r"(temp[3]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[3]) : "r"(temp[3]), "r"(temp[4]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[4]) : "r"(temp[4]), "r"(temp[5]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[5]) : "r"(temp[5]), "r"(temp[6]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[6]) : "r"(temp[6]), "r"(temp[7]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[7]) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));

		for (int k = 0; k < 9; k++) {
			((uint32_t *)B0)[(k + qbuf) & 63] = temp[k];
		}


	}

#if __CUDA_ARCH__ < 500
	Blake2S(input, input, key);
#else
	Blake2S_v2(input, input, key);
#endif
	bufidx = 0;
	bufhelper = ((uchar4*)input)[0];
	for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
	bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;
	qbuf = bufidx / 4;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;


	for (int k = 7; k < 9; k++) {
		temp[k] = ((uint32_t *)B0)[(k + qbuf) % 64];
	}
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(output) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));
	output ^= input[7];
	output ^= cdata7;

}


static __forceinline__ __device__ void fastkdf32_v3(int thread, const  uint32_t  nonce, const uint32_t * __restrict__ salt, const uint32_t * __restrict__  s_data, uint32_t &output)
{

	uint32_t temp[9];


	uint8_t bufidx = 0;
	uchar4 bufhelper;

	//	uint32_t temp[9];

	//	uint32_t  B0[64];
#define Bshift 16*thread

	uint32_t* B0 = (uint32_t*)&B2[Bshift];
	uint32_t cdata7 = s_data[7];
	uint32_t data18 = s_data[18];
	uint32_t data20 = s_data[0];


	((uintx64*)B0)[0] = ((uintx64*)salt)[0];
//	((ulonglong4*)B0)[8] = ((ulonglong4*)salt)[0];
	uint32_t input[BLAKE2S_BLOCK_SIZE / 4]; uint32_t key[BLAKE2S_BLOCK_SIZE / 4] = { 0 };
	((uint816*)input)[0] = ((uint816*)s_data)[0];
	((uint48*)key)[0] = ((uint48*)salt)[0];
	uint32_t qbuf, rbuf, bitbuf;

#pragma nounroll
	for (int i = 0; i < 31; i++)
	{
#if __CUDA_ARCH__ < 500
		Blake2S(input, input, key);
#else
		Blake2S_v2(input, input, key);
#endif
		bufidx = 0;
		bufhelper = ((uchar4*)input)[0];
		for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
		bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;
		qbuf = bufidx / 4;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;
		uint32_t shifted[9];

		shift256R4(shifted, ((uint8*)input)[0], bitbuf);


		for (int k = 0; k < 9; k++) {
			temp[k] = __ldg(&((uint32_t*)B0)[(k + qbuf) % 64]);
		}

		((uint28*)temp)[0] ^= ((uint28*)shifted)[0];
		temp[8] ^= shifted[8];



		uint32_t a = s_data[qbuf % 64], b;
		//#pragma unroll
		for (int k = 0; k < 8; k++) {
			b = s_data[(qbuf + 2 * k + 1) % 64];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[2 * k]) : "r"(a), "r"(b), "r"(bitbuf));
			a = s_data[(qbuf + 2 * k + 2) % 64];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[2 * k + 1]) : "r"(b), "r"(a), "r"(bitbuf));
		}


		uint32_t noncepos = 19 - qbuf % 20;
		if (noncepos <= 16 && qbuf < 60) {
			if (noncepos != 0)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos - 1]) : "r"(data18), "r"(nonce), "r"(bitbuf));
			if (noncepos != 16)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos]) : "r"(nonce), "r"(data20), "r"(bitbuf));
		}

		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[0]) : "r"(temp[0]), "r"(temp[1]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[1]) : "r"(temp[1]), "r"(temp[2]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[2]) : "r"(temp[2]), "r"(temp[3]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[3]) : "r"(temp[3]), "r"(temp[4]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[4]) : "r"(temp[4]), "r"(temp[5]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[5]) : "r"(temp[5]), "r"(temp[6]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[6]) : "r"(temp[6]), "r"(temp[7]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[7]) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));

		for (int k = 0; k < 9; k++) {
			((uint32_t*)B0)[(k + qbuf) % 64] = temp[k];
		}

//		}
	}

#if __CUDA_ARCH__ < 500
	Blake2S(input, input, key);
#else
	Blake2S_v2(input, input, key);
#endif
	bufidx = 0;
	bufhelper = ((uchar4*)input)[0];
	for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
	bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;
	qbuf = bufidx / 4;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;



	temp[7] = __ldg(&B0[(qbuf + 7) % 64]);
	temp[8] = __ldg(&B0[(qbuf + 8) % 64]);
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(output) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));
	output ^= input[7];
	output ^= cdata7;

}




#if CUDART_VERSION >= 7000
#define SHIFT 128
#define TPB 128
#else
#define SHIFT 128
#define TPB 64
#endif
#define TPB2 128



__global__ __launch_bounds__(TPB2, 1) void neoscrypt_gpu_hash_start(int stratum, int threads, uint32_t startNonce)
{
	__shared__ uint32_t s_data[64];

	if (threadIdx.x < 64)
		s_data[threadIdx.x] = c_data[threadIdx.x];
//		for (int i = 0; i<2; i++) {
//	s_data[i+2*threadIdx.x] = c_data[i+2*threadIdx.x];

//}
	__syncthreads();
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint32_t nonce = startNonce + thread;

	uint32_t ZNonce = (stratum) ? cuda_swab32(nonce) : nonce; //freaking morons !!!

#if __CUDA_ARCH__ < 500
	fastkdf256_v1(thread, ZNonce, s_data);
#else
	fastkdf256_v2(thread, ZNonce, s_data);
#endif

}

__global__ __launch_bounds__(TPB, 1) void neoscrypt_gpu_hash_chacha1_stream1(int threads, uint32_t startNonce)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	int shift = SHIFT * 8 * thread;
	unsigned int shiftTr = 8 * thread;


	vectypeS X[8];
	for (int i = 0; i < 8; i++)
		X[i] = __ldg4(&(Input + shiftTr)[i]);


#pragma nounroll
	for (int i = 0; i < 128; ++i)
	{
		uint32_t offset = shift + i * 8;
		for (int j = 0; j < 8; j++)
			(W + offset)[j] = X[j];
		neoscrypt_chacha((uint16*)X);

	}
	for (int i = 0; i < 8; i++)
		(Tr + shiftTr)[i] = X[i];

}

__global__ __launch_bounds__(TPB, 1) void neoscrypt_gpu_hash_chacha2_stream1(int threads, uint32_t startNonce)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	int shift = SHIFT * 8 * thread;
	int shiftTr = 8 * thread;

	vectypeS X[8];
	for (int i = 0; i < 8; i++)
		X[i] = __ldg4(&(Tr + shiftTr)[i]);

#pragma nounroll
	for (int t = 0; t < 128; t++)
	{
		int idx = (X[6].x.x & 0x7F) << 3;

		for (int j = 0; j < 8; j++)
			X[j] ^= __ldg4(&(W + shift + idx)[j]);
		neoscrypt_chacha((uint16*)X);

	}

	for (int i = 0; i < 8; i++)
		(Tr + shiftTr)[i] = X[i];  // best checked

}

__global__ __launch_bounds__(TPB, 1) void neoscrypt_gpu_hash_salsa1_stream1_orig(int threads, uint32_t startNonce)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);

	int shift = SHIFT * 8 * thread;
	int shiftTr = 8 * thread;

	vectypeS Z[8];

	#pragma unroll
	for (int i = 0; i < 8; i++)
		Z[i] = __ldg4(&(Input + shiftTr)[i]);

// #pragma nounroll
	#pragma unroll
	for (int i = 0; i < 128; ++i)
	{
		for (int j = 0; j < 8; j++)
			(W2 + shift + i * 8)[j] = Z[j];
		neoscrypt_salsa((uint16*)Z);
	}

	#pragma unroll
	for (int i = 0; i < 8; i++)
		(Tr2 + shiftTr)[i] = Z[i];
}

static __device__ __inline__ void __copy16(uint4 *dest, const uint4 *src)
{
	// uint4 a = {1,2,3,4};
	// uint4 b;

	// asm("ld.local.cs.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(b.x), "=r"(b.y), "=r"(b.z), "=r"(b.w) : "l"(&(a.x)));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(b.x), "=r"(b.y), "=r"(b.z), "=r"(b.w) : "l"(&(a.x)));

	// asm volatile ("{\n\t"
	// 	".reg .u32 a,b,c,d; \n\t"
	// 	"ld.global.nc.v4.u32 {a,b,c,d}, [%1]; \n\t"
	// 	"mov.u32 [%0], a; \n\t"
	// 	"mov.u32 [%0+1], b; \n\t"
	// 	"mov.u32 [%0+2], c; \n\t"
	// 	"mov.u32 [%0+3], d; \n\t"
	// 	"}"
	// 	: "=l"((uint*)dest) : "l"(src));
	// uint* sptr = (uint*)src;
	// uint* dptr = (uint*)dest;
	uint* sptr = (uint*)src;
	uint64_t* dptr = (uint64_t*)dest;

	// asm volatile ("{\n\t"
	// 	".reg .u64 a,b; \n\t"
	// 	"ld.global.nc.v2.u64 {a,b}, [%1]; \n\t"
	// 	"st.local.cs.v2.u64 [%0], {a,b}; \n\t"
	// 	"}"
	// : "=l"(dptr) : "l"(sptr) );

	asm("ld.global.nc.v2.u64 {%0,%1}, [%2];"  : "=l"(dptr[0]), "=l"(dptr[1]) : "l"(src+0));
	asm("ld.global.nc.v2.u64 {%0,%1}, [%2];"  : "=l"(dptr[2]), "=l"(dptr[3]) : "l"(src+1));

	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[0].x), "=r"(dest[0].y), "=r"(dest[0].z), "=r"(dest[0].w) : "l"(src+0));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[1].x), "=r"(dest[1].y), "=r"(dest[1].z), "=r"(dest[1].w) : "l"(src+1));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[2].x), "=r"(dest[2].y), "=r"(dest[2].z), "=r"(dest[2].w) : "l"(src+2));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[3].x), "=r"(dest[3].y), "=r"(dest[3].z), "=r"(dest[3].w) : "l"(src+3));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[4].x), "=r"(dest[4].y), "=r"(dest[4].z), "=r"(dest[4].w) : "l"(src+4));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[5].x), "=r"(dest[5].y), "=r"(dest[5].z), "=r"(dest[5].w) : "l"(src+5));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[6].x), "=r"(dest[6].y), "=r"(dest[6].z), "=r"(dest[6].w) : "l"(src+6));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[7].x), "=r"(dest[7].y), "=r"(dest[7].z), "=r"(dest[7].w) : "l"(src+7));

	dest+=2;
	src+=2;

	for(int i=2;i<16;++i)
		(*dest++) = (*src++);

	// dest[0] = src[0];
	// dest[1] = src[1];
	// dest[2] = src[2];
	// dest[3] = src[3];
	// dest[4] = src[4];
	// dest[5] = src[5];
	// dest[6] = src[6];
	// dest[7] = src[7];

	// uint28 ret;
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[0].x.x), "=r"(dest[0].x.y), "=r"(dest[0].y.x), "=r"(dest[0].y.y) : __LDG_PTR(src));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4+16];" : "=r"(dest[0].z.x), "=r"(dest[0].z.y), "=r"(dest[0].w.x), "=r"(dest[0].w.y) : __LDG_PTR(src));
	// dest[0] = ret;

	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[1].x.x), "=r"(dest[1].x.y), "=r"(dest[1].y.x), "=r"(dest[1].y.y) : __LDG_PTR(src+1));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4+16];" : "=r"(dest[1].z.x), "=r"(dest[1].z.y), "=r"(dest[1].w.x), "=r"(dest[1].w.y) : __LDG_PTR(src+1));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[2].x.x), "=r"(dest[2].x.y), "=r"(dest[2].y.x), "=r"(dest[2].y.y) : __LDG_PTR(src+2));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4+16];" : "=r"(dest[2].z.x), "=r"(dest[2].z.y), "=r"(dest[2].w.x), "=r"(dest[2].w.y) : __LDG_PTR(src+2));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[3].x.x), "=r"(dest[3].x.y), "=r"(dest[3].y.x), "=r"(dest[3].y.y) : __LDG_PTR(src+3));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4+16];" : "=r"(dest[3].z.x), "=r"(dest[3].z.y), "=r"(dest[3].w.x), "=r"(dest[3].w.y) : __LDG_PTR(src+3));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[4].x.x), "=r"(dest[4].x.y), "=r"(dest[4].y.x), "=r"(dest[4].y.y) : __LDG_PTR(src+4));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4+16];" : "=r"(dest[4].z.x), "=r"(dest[4].z.y), "=r"(dest[4].w.x), "=r"(dest[4].w.y) : __LDG_PTR(src+4));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[5].x.x), "=r"(dest[5].x.y), "=r"(dest[5].y.x), "=r"(dest[5].y.y) : __LDG_PTR(src+5));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4+16];" : "=r"(dest[5].z.x), "=r"(dest[5].z.y), "=r"(dest[5].w.x), "=r"(dest[5].w.y) : __LDG_PTR(src+5));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[6].x.x), "=r"(dest[6].x.y), "=r"(dest[6].y.x), "=r"(dest[6].y.y) : __LDG_PTR(src+6));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4+16];" : "=r"(dest[6].z.x), "=r"(dest[6].z.y), "=r"(dest[6].w.x), "=r"(dest[6].w.y) : __LDG_PTR(src+6));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4];"  : "=r"(dest[7].x.x), "=r"(dest[7].x.y), "=r"(dest[7].y.x), "=r"(dest[7].y.y) : __LDG_PTR(src+7));
	// asm("ld.global.nc.v4.u32 {%0,%1,%2,%3}, [%4+16];" : "=r"(dest[7].z.x), "=r"(dest[7].z.y), "=r"(dest[7].w.x), "=r"(dest[7].w.y) : __LDG_PTR(src+7));

// 	return ret;
}

__global__ __launch_bounds__(TPB, 1) void neoscrypt_gpu_hash_salsa1_stream1(int threads, uint32_t startNonce)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);

	// Now we try to process 2 of those blocks at the same time:
	int shift = SHIFT * 8 * thread;
	int shiftTr = 8 * thread;

	uint4 Z[16];

	__copy16((uint4*)Z,(uint4*)(Input+shiftTr));

	// #pragma unroll
	// for (int i = 0; i < 8; i++)
	// 	Z[i] = (Input + shiftTr)[i];

// #pragma nounroll
	uint4* ptr = (uint4*)(W2 + shift);

	#pragma unroll
	for (int i = 0; i < 128; ++i)
	{
		#pragma unroll
		for (int j = 0; j < 16; j++)
			(*ptr++) = Z[j];
		// __copy16(ptr,Z);
		// ptr += 8;

		neoscrypt_salsa((uint16*)Z);
	}

	// __copy16((uint4*)(Tr2+shiftTr),(uint4*)Z);

	ptr = (uint4*)(Tr2 + shiftTr);
	#pragma unroll
	for (int i = 0; i < 16; i++)
		(*ptr++) = Z[i];
}

#define BSIZE 32

// __launch_bounds__(TPB, 1)

__global__  void neoscrypt_gpu_hash_salsa1_stream1_opt(int threads, uint32_t startNonce, unsigned long long* time)
{
	// unsigned long long startTime = clock();

	int ioffset = BSIZE * 64 * blockIdx.x;
	int woffset = BSIZE * SHIFT * 64 * blockIdx.x;

	int x = threadIdx.x;

	// Input is provided as uint28 pointer, each uint28 is 2*uint4 = 8 uint
	// so if the stride is 8 uint28, then it is 8x8=64 uint
	// Same result for the shift: if it is 8 uint28, then this is 64 uint:

	// int shiftTr = 64 * offset;
	// int shift = SHIFT * 64 * offset;
	
	uint* iPtr = ((uint*)Input)+ioffset;

	// Prepare the buffer containing all the input rows:
	// Z rows contain 8 uint28, and thus 64 uint, to avoid memory bank conflits
	// We add 1 to this size:
	__shared__ uint Z[BSIZE][64+1];

	// Fill the input array:
	for(int j=0;j<BSIZE;++j)
	{
		Z[j][x] = iPtr[j*64 + x];
		Z[j][32+x] = iPtr[j*64 + 32 + x];
	}

	// Need to synchronize the threads:
	// __syncthreads();

	// #pragma nounroll
	uint* dPtr = ((uint*)W2) + woffset;

	#pragma unroll
	for (int i = 0; i < 128; ++i)
	{
		for(int j=0;j<BSIZE;++j)
		{
			dPtr[i*64 + j*SHIFT*64 + x] = Z[j][x];
			dPtr[i*64 + j*SHIFT*64 + 32 + x] = Z[j][32 + x];
		}
		// __syncthreads();

		// #pragma unroll
		// for (int j = 0; j < 16; j++)
		// 	(*ptr++) = Z[j];
		// __copy16(ptr,Z);
		// ptr += 8;

		neoscrypt_salsa((uint16*)Z[x]);
	}

	// Copy the final data in to the Tr2 buffer:
	dPtr = ((uint*)Tr2)+ioffset;
	for(int j=0;j<BSIZE;++j)
	{
		dPtr[j*64 + x] = Z[j][x];
		dPtr[j*64 + 32 + x] = Z[j][32 + x];
	}

	// No need to sync the threads here: we are done.
	// __syncthreads();

	// unsigned long long endTime = clock();
	// *time = (endTime - startTime);
}

__global__ __launch_bounds__(TPB, 1) void neoscrypt_gpu_hash_salsa2_stream1(int threads, uint32_t startNonce)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	int shift = SHIFT * 8 * thread;
	int shiftTr = 8 * thread;

	vectypeS X[8];
	for (int i = 0; i < 8; i++)
		X[i] = __ldg4(&(Tr2 + shiftTr)[i]);

#pragma nounroll
	for (int t = 0; t < 128; t++)
	{
		int idx = (X[6].x.x & 0x7F) << 3;

		for (int j = 0; j < 8; j++)
			X[j] ^= __ldg4(&(W2 + shift + idx)[j]);
		neoscrypt_salsa((uint16*)X);

	}
	for (int i = 0; i < 8; i++)
		(Tr2 + shiftTr)[i] = X[i];  // best checked

}

__global__  __launch_bounds__(TPB, 1) void neoscrypt_gpu_hash_salsa1_stream1_merge(int threads, uint32_t startNonce)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);

	int shift = SHIFT * 8 * thread;
	int shiftTr = 8 * thread;
	int x = threadIdx.x;

	vectypeS Z[8];

	#pragma unroll
	for (int i = 0; i < 8; i++)
		Z[i] = __ldg4(&(Input + shiftTr)[i]);

// #pragma nounroll
	#pragma unroll
	for (int i = 0; i < 128; ++i)
	{
		for (int j = 0; j < 8; j++)
			(W2 + shift + i * 8)[j] = Z[j];
		neoscrypt_salsa((uint16*)Z);
	}

	#pragma unroll
	for (int t = 0; t < 128; t++)
	{
		int idx = (Z[6].x.x & 0x7F) << 3;

		for (int j = 0; j < 8; j++)
			Z[j] ^= __ldg4(&(W2 + shift + idx)[j]);
		neoscrypt_salsa((uint16*)Z);
	}

	for (int i = 0; i < 8; i++)
		(Tr2 + shiftTr)[i] = Z[i];  // best checked
}



__global__ __launch_bounds__(TPB2, 1) void neoscrypt_gpu_hash_ending(int stratum, int threads, uint32_t startNonce, uint32_t *nonceVector)
{
	__shared__ uint32_t s_data[64];
	/*
		if (threadIdx.x<40)
			for (int i = 0; i<2; i++)
	       s_data[i + 2 * threadIdx.x] = c_data[i + 2 * threadIdx.x];
	*/
	if (threadIdx.x < 64)
		s_data[threadIdx.x] = c_data[threadIdx.x];
	__syncthreads();
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint32_t nonce = startNonce + thread;

	int shiftTr = 8 * thread;
	vectypeS Z[8];
	uint32_t outbuf;

	uint32_t ZNonce = (stratum) ? cuda_swab32(nonce) : nonce;

//		for (int i = 0; i<8; i++)
//		Z[i] = __ldg4(&(Tr + shiftTr)[i]);
	for (int i = 0; i < 8; i++)
		Z[i] = __ldg4(&(Tr2 + shiftTr)[i]) ^ __ldg4(&(Tr + shiftTr)[i]);
#if __CUDA_ARCH__ < 500
	fastkdf32_v1(thread, ZNonce, (uint32_t*)Z, s_data, outbuf);
#else
	fastkdf32_v3(thread, ZNonce, (uint32_t*)Z, s_data, outbuf);
#endif
	if (outbuf <= pTarget[7]) {
		uint32_t tmp = atomicExch(&nonceVector[0], nonce);
	}
}


void neoscrypt_cpu_init_2stream(int thr_id, int threads, uint32_t *hash, uint32_t *hash2, uint32_t *Trans1, uint32_t *Trans2, uint32_t *Trans3, uint32_t *Bhash)
{
	hipMemcpyToSymbol(HIP_SYMBOL(B2), &Bhash, sizeof(Bhash), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(W), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(W2), &hash2, sizeof(hash2), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(Tr), &Trans1, sizeof(Trans1), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(Tr2), &Trans2, sizeof(Trans2), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(Input), &Trans3, sizeof(Trans3), 0, hipMemcpyHostToDevice);

	hipMalloc(&d_NNonce[thr_id], sizeof(uint32_t));
	hipMalloc(&d_time[thr_id], sizeof(unsigned long long));

	// Create the streams:
	hipStreamCreate(&g_stream[thr_id*2]);
	hipStreamCreate(&g_stream[thr_id*2+1]);

}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__host__ uint32_t neoscrypt_cpu_hash_k4_2stream(int stratum, int thr_id, int threads, uint32_t startNounce, int order, unsigned long long &tres)
{
	uint32_t result[MAX_GPUS] = { 0xffffffff };
	hipMemset(d_NNonce[thr_id], 0xffffffff, sizeof(uint32_t));


	const int threadsperblock = TPB;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	const int threadsperblock2 = TPB2;
	dim3 grid2((threads + threadsperblock2 - 1) / threadsperblock2);
	dim3 block2(threadsperblock2);

	const int threadsperblock3 = BSIZE;
	dim3 grid3((threads + threadsperblock3 - 1) / threadsperblock3);
	dim3 block3(threadsperblock3);

	//	neoscrypt_gpu_hash_orig << <grid, block >> >(threads, startNounce, d_NNonce[thr_id]);

	neoscrypt_gpu_hash_start << <grid2, block2, 0, g_stream[thr_id*2] >> >(stratum, threads, startNounce); //fastkdf

	hipDeviceSynchronize();

	neoscrypt_gpu_hash_chacha1_stream1 << <grid, block, 0, g_stream[thr_id*2] >> >(threads, startNounce); //salsa
	gpuErrchk( hipPeekAtLastError() );
	neoscrypt_gpu_hash_chacha2_stream1 << <grid, block, 0, g_stream[thr_id*2] >> >(threads, startNounce); //salsa
	gpuErrchk( hipPeekAtLastError() );

	// neoscrypt_gpu_hash_salsa1_stream1_merge << <grid, block, 0, g_stream[thr_id*2+1] >> >(threads, startNounce); //chacha
	// neoscrypt_gpu_hash_salsa1_stream1 << <grid, block, 0, g_stream[thr_id*2+1] >> >(threads, startNounce); //chacha
	neoscrypt_gpu_hash_salsa1_stream1_orig << <grid, block, 0, g_stream[thr_id*2+1] >> >(threads, startNounce); //chacha
	gpuErrchk( hipPeekAtLastError() );
	// neoscrypt_gpu_hash_salsa1_stream1_opt << <grid3, block3, 0, g_stream[thr_id*2+1] >> >(threads, startNounce, d_time[thr_id]); //chacha
	neoscrypt_gpu_hash_salsa2_stream1 << <grid, block, 0, g_stream[thr_id*2+1] >> >(threads, startNounce); //chacha
	gpuErrchk( hipPeekAtLastError() );

	hipDeviceSynchronize();
	// hipStreamDestroy(g_stream[thr_id*2+1]); //will do the synchronization
	neoscrypt_gpu_hash_ending << <grid2, block2, 0, g_stream[thr_id*2] >> >(stratum, threads, startNounce, d_NNonce[thr_id]); //fastkdf+end


	MyStreamSynchronize(NULL, order, thr_id);
	hipMemcpy(&result[thr_id], d_NNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

	hipMemcpy(&tres, d_time[thr_id], sizeof(unsigned long long), hipMemcpyDeviceToHost);

	// hipStreamDestroy(g_stream[thr_id*2]);


	return result[thr_id];
}

__host__ void neoscrypt_setBlockTarget(uint32_t* pdata, const void *target)
{

	unsigned int PaddedMessage[80]; //brings balance to the force
	uint32_t input[16], key[16] = {0};

	for (int i = 0; i < 20; i++) PaddedMessage[i     ] = pdata[i];
	for (int i = 0; i < 20; i++) PaddedMessage[i + 20] = pdata[i];
	for (int i = 0; i < 20; i++) PaddedMessage[i + 40] = pdata[i];
	for (int i = 0; i < 4; i++)  PaddedMessage[i + 60] = pdata[i];
	for (int i = 0; i < 16; i++) PaddedMessage[i + 64] = pdata[i];
	PaddedMessage[19] = 0;
	PaddedMessage[39] = 0;
	PaddedMessage[59] = 0;

	((uint16*)input)[0] = ((uint16*)pdata)[0];
	((uint8*)key)[0] = ((uint8*)pdata)[0];
//		for (int i = 0; i<10; i++) { printf(" pdata/input %d %08x %08x \n",i,pdata[2*i],pdata[2*i+1]); }


	Blake2Shost(input, key);


	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), target, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(input_init), input, 16 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(key_init), key, 16 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(c_data), PaddedMessage, 40 * sizeof(uint64_t), 0, hipMemcpyHostToDevice);
//		hipMemcpyToSymbol(HIP_SYMBOL(c_data2), PaddedMessage, 40 * sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

